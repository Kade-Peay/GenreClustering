#include "hip/hip_runtime.h"
#include "utils.hpp"
#include <ctime>
#include <fstream>
#include <iostream>
#include <sstream>
#include <vector>
#include <cfloat>
#include <string>
#include <hip/hip_runtime.h>

__global__ void AssignToCluster(Point* points, Point* centroids, int clusterId) {
    int idx = threadIdx.x;

    Point p = points[idx];

    double dist = (p.x - centroids[clusterId].x) * (p.x - centroids[clusterId].x) +
                  (p.y - centroids[clusterId].y) * (p.y - centroids[clusterId].y);

    if (dist < p.minDist)
    {
        p.minDist = dist;
        p.cluster = clusterId;
    }
    points[idx] = p;
}

__global__ void ResetDistance(Point* points){
    int idx = threadIdx.x;
    points[idx].minDist = DBL_MAX;
}

int main(int argc, char* argv[])
{
    if (argc != 4) {
        std::cerr << "Usage: " << argv[0] << " <input_file> <number_of_clusters> <threads_per_block>" << std::endl;
        return -1;
    }

    std::string inputFile = argv[1];
    int k = std::stoi(argv[2]);
    int threadsPerBlock = std::stoi(argv[3]);

    std::vector<Point> points = readcsv(inputFile);

    if (points.empty())
    {
        std::cerr << "No data points loaded. Check your input file.\n";
        return 1;
    }

    int epochs = 100; // number of iterations

    std::vector<Point> centroids;
    srand(time(0));

    // Initialize centroids with random points
    for (int i = 0; i < k; ++i)
    {
        centroids.push_back(points.at(rand() % points.size()));
    }

    // Allocate device memory
    Point* d_points;
    Point* d_centroids;
    hipMalloc(&d_points, points.size() * sizeof(Point));
    hipMalloc(&d_centroids, k * sizeof(Point));

    // Copy data to device
    hipMemcpy(d_points, points.data(), points.size() * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, centroids.data(), k * sizeof(Point), hipMemcpyHostToDevice);

    int blocks = (k + threadsPerBlock - 1) / threadsPerBlock;

    for (int epoch = 0; epoch < epochs; ++epoch)
    {
        // Assign points to clusters
        for (int i = 0; i < centroids.size(); ++i)
        {
            AssignToCluster<<<blocks, threadsPerBlock>>>(d_points, d_centroids, i);
        }
        hipDeviceSynchronize();

        std::vector<int> nPoints(k, 0);
        std::vector<double> sumX(k, 0.0);
        std::vector<double> sumY(k, 0.0);

        // Accumulate points for new centroids
        hipMemcpy(points.data(), d_points, points.size() * sizeof(Point), hipMemcpyDeviceToHost);
        for (auto it = points.begin(); it != points.end(); ++it)
        {
            int clusterId = it->cluster;
            nPoints[clusterId] += 1;
            sumX[clusterId] += it->x;
            sumY[clusterId] += it->y;
        }

        // reset distance
        ResetDistance<<<blocks, threadsPerBlock>>>(d_points);
        hipDeviceSynchronize();

        // Compute new centroids
        hipMemcpy(centroids.data(), d_centroids, k * sizeof(Point), hipMemcpyDeviceToHost);
        for (auto c = begin(centroids); c != end(centroids); ++c)
        {
            int clusterId = c - begin(centroids);
            if (nPoints[clusterId] != 0)
            {
                c->x = sumX[clusterId] / nPoints[clusterId];
                c->y = sumY[clusterId] / nPoints[clusterId];
            }
        }
        hipMemcpy(d_centroids, centroids.data(), k * sizeof(Point), hipMemcpyHostToDevice);
    }

    // Clean up
    hipFree(d_points);
    hipFree(d_centroids);

    // Write results to output file
    std::ofstream myfile("output.csv");
    myfile << "danceability,energy,cluster\n";

    for (const auto &point : points)
    {
        myfile << point.x << "," << point.y << "," << point.cluster << "\n";
    }
    myfile.close();

    std::cout << "Clustering complete. Results saved to output.csv\n";
    return 0;
}
