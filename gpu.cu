#include "hip/hip_runtime.h"
#include "utils.hpp"
#include <hip/hip_runtime.h>
#include <cfloat>
#include <ctime>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

__global__ void AssignToCluster(Point* points, Point* centroids, int k, int numPoints) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numPoints) return;

    Point p = points[idx];
    p.minDist = DBL_MAX;

    for (int clusterId = 0; clusterId < k; clusterId++)
    {
        double dist = 
            (p.danceability - centroids[clusterId].danceability) * (p.danceability - centroids[clusterId].danceability) +
            (p.valence - centroids[clusterId].valence) * (p.valence - centroids[clusterId].valence) + 
            (p.energy - centroids[clusterId].energy) * (p.energy - centroids[clusterId].energy);

        if (dist < p.minDist)
        {
            p.minDist = dist;
            p.cluster = clusterId;
        }
    }
    points[idx] = p;
}

extern "C" void Malloc(Point** points, size_t size){
    hipError_t err = hipMalloc(points, size * sizeof(Point));
    if (err != hipSuccess) {
        std::cerr << "hipMalloc failed: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
}

extern "C" void MemcpyHost(Point* devicePoints, Point* hostPoints, size_t size){
    hipError_t err = hipMemcpy(devicePoints, hostPoints, size * sizeof(Point), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "hipMalloc failed: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
}

extern "C" void MemcpyDevice(Point* devicePoints, Point* hostPoints, size_t size){
    hipError_t err = hipMemcpy(hostPoints, devicePoints, size * sizeof(Point), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "hipMalloc failed: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
}

extern "C" void Free(Point* points){
    hipError_t err = hipFree(points);
    if (err != hipSuccess) {
        std::cerr << "hipMalloc failed: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
}

extern "C" float AssignToCluster(int blocks, int threadsPerBlock, Point* points, Point* centroids, int k, int numPoints) {
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);

    hipEventRecord(start1);
    AssignToCluster<<<blocks, threadsPerBlock>>>(points, centroids, k, numPoints);
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start1, stop1);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Kernel failed: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    return milliseconds;
}
