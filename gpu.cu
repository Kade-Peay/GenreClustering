#include "hip/hip_runtime.h"
#include "utils.hpp"
#include <hip/hip_runtime.h>
#include <cfloat>
#include <ctime>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

__global__ void AssignToCluster(Point* points, Point* centroids, int k) {
    int idx = threadIdx.x;

    Point p = points[idx];
    p.minDist = DBL_MAX;

    for (int clusterId = 0; clusterId < k; clusterId++)
    {
        double dist = 
            (p.danceability - centroids[clusterId].danceability) * (p.danceability - centroids[clusterId].danceability) +
            (p.valence - centroids[clusterId].valence) * (p.valence - centroids[clusterId].valence) + 
            (p.energy - centroids[clusterId].energy) * (p.energy - centroids[clusterId].energy);

        if (dist < p.minDist)
        {
            p.minDist = dist;
            p.cluster = clusterId;
        }
    }
    points[idx] = p;
}

extern "C" void Malloc(Point** points, int size){
    hipMalloc(points, size);
}

extern "C" void MemcpyHost(Point* devicePoints, Point* hostPoints, int size){
    hipMemcpy(devicePoints, hostPoints, size * sizeof(Point), hipMemcpyHostToDevice);
}

extern "C" void MemcpyDevice(Point* devicePoints, Point* hostPoints, int size){
    hipMemcpy(hostPoints, devicePoints, size * sizeof(Point), hipMemcpyDeviceToHost);

}

extern "C" void Free(Point* points){
    hipFree(points);
}

extern "C" void AssignToCluster(int blocks, int threadsPerBlock, Point* points, Point* centroids, int k) {
    AssignToCluster<<<blocks, threadsPerBlock>>>(points, centroids, k);
    hipDeviceSynchronize();
}
